#include "hip/hip_runtime.h"
/*
Kernels for layernorm backward pass.

Compile example:
nvcc -O3 --use_fast_math layernorm_backward.cu -o layernorm_backward

version 1 is naive port from CPU code to kernel: parallelizes over B,T, loops over C
./layernorm_backward 1

version 2 moves a lot of reduction to shared memory over global memory
./layernorm_backward 2
*/

#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
#include "common.h"

// ----------------------------------------------------------------------------
// CPU code reference

void layernorm_forward_cpu(float* out, float* mean, float* rstd,
                       const float* inp, const float* weight, const float* bias,
                       int B, int T, int C) {
    // reference: https://pytorch.org/docs/stable/generated/torch.nn.LayerNorm.html
    // both inp and out are (B,T,C) of the activations
    // mean and rstd are (B,T) buffers, to be used later in backward pass
    // at each position (b,t) of the input, the C-dimensional vector
    // of activations gets normalized, then scaled and shifted
    float eps = 1e-5f;
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            // seek to the input position inp[b,t,:]
            const float* x = inp + b * T * C + t * C;
            // calculate the mean
            float m = 0.0f;
            for (int i = 0; i < C; i++) {
                m += x[i];
            }
            m = m/C;
            // calculate the variance (without any bias correction)
            float v = 0.0f;
            for (int i = 0; i < C; i++) {
                float xshift = x[i] - m;
                v += xshift * xshift;
            }
            v = v/C;
            // calculate the rstd (reciprocal standard deviation)
            float s = 1.0f / sqrtf(v + eps);
            // seek to the output position in out[b,t,:]
            float* out_bt = out + b * T * C + t * C;
            for (int i = 0; i < C; i++) {
                float n = (s * (x[i] - m)); // normalize
                float o = n * weight[i] + bias[i]; // scale and shift
                out_bt[i] = o; // write
            }
            // cache the mean and rstd for the backward pass later
            mean[b * T + t] = m;
            rstd[b * T + t] = s;
        }
    }
}

void layernorm_backward_cpu(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    for (int b = 0; b < B; b++) {
        for (int t = 0; t < T; t++) {
            const float* dout_bt = dout + b * T * C + t * C;
            const float* inp_bt = inp + b * T * C + t * C;
            float* dinp_bt = dinp + b * T * C + t * C;
            const float mean_bt = mean[b * T + t];
            const float rstd_bt = rstd[b * T + t];

            // first: two reduce operations
            float dnorm_mean = 0.0f;
            float dnorm_norm_mean = 0.0f;
            for (int i = 0; i < C; i++) {
                float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                dnorm_mean += dnorm_i;
                dnorm_norm_mean += dnorm_i * norm_bti;
            }
            dnorm_mean = dnorm_mean / C;
            dnorm_norm_mean = dnorm_norm_mean / C;

            // now iterate again and accumulate all the gradients
            for (int i = 0; i < C; i++) {
                float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
                float dnorm_i = weight[i] * dout_bt[i];
                // gradient contribution to bias
                dbias[i] += dout_bt[i];
                // gradient contribution to weight
                dweight[i] += norm_bti * dout_bt[i];
                // gradient contribution to input
                float dval = 0.0f;
                dval += dnorm_i; // term 1
                dval -= dnorm_mean; // term 2
                dval -= norm_bti * dnorm_norm_mean; // term 3
                dval *= rstd_bt; // final scale
                dinp_bt[i] += dval;
            }
        }
    }
}

// ----------------------------------------------------------------------------
// GPU kernels

// super naive kernel that just parallelizes over B,T and loops over C
__global__ void layernorm_backward_kernel1(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= B*T) return;
    int b = idx / T;
    int t = idx % T;

    const float* dout_bt = dout + b * T * C + t * C;
    const float* inp_bt = inp + b * T * C + t * C;
    float* dinp_bt = dinp + b * T * C + t * C;
    const float mean_bt = mean[b * T + t];
    const float rstd_bt = rstd[b * T + t];

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = 0; i < C; i++) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = 0; i < C; i++) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias[i], dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight[i], norm_bti * dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] += dval;
    }
}

// uses shared memory instead for the reduces
__global__ void layernorm_backward_kernel2(float* dinp, float* dweight, float* dbias,
                                           const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                                           int B, int T, int C) {
    extern __shared__ float shared[]; // size = 2 * C

    namespace cg = cooperative_groups;
    cg::thread_block block = cg::this_thread_block();
    cg::thread_block_tile<32> warp = cg::tiled_partition<32>(block);
    int idx = blockIdx.x * warp.meta_group_size() + warp.meta_group_rank();
    int N = B * T;
    if(idx >= N) { return; } // thread guards

    int b = idx / T;
    int t = idx % T;

    const float* dout_bt = dout + b * T * C + t * C;
    const float* inp_bt = inp + b * T * C + t * C;
    float* dinp_bt = dinp + b * T * C + t * C;
    const float mean_bt = mean[b * T + t];
    const float rstd_bt = rstd[b * T + t];

    // the first half of shared memory is bias, second is weight
    float* dbias_shared = shared;
    float* dweight_shared = shared + C;

    // init shared memory to zero
    #pragma unroll
	for(int i = threadIdx.x; i < C; i+= blockDim.x){
       dbias_shared[i] = 0.0f;
       dweight_shared[i] = 0.0f;
    }
    __syncthreads();

    // first: two reduce operations
    float dnorm_mean = 0.0f;
    float dnorm_norm_mean = 0.0f;
    for (int i = warp.thread_rank(); i < C; i  += warp.size()) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        dnorm_mean += dnorm_i;
        dnorm_norm_mean += dnorm_i * norm_bti;
    }
    dnorm_mean = cg::reduce(warp, dnorm_mean, cg::plus<float>{});
    dnorm_norm_mean = cg::reduce(warp, dnorm_norm_mean, cg::plus<float>{});
    dnorm_mean = dnorm_mean / C;
    dnorm_norm_mean = dnorm_norm_mean / C;

    // now iterate again and accumulate all the gradients
    for (int i = warp.thread_rank(); i < C; i += warp.size()) {
        float norm_bti = (inp_bt[i] - mean_bt) * rstd_bt;
        float dnorm_i = weight[i] * dout_bt[i];
        // gradient contribution to bias
        atomicAdd(&dbias_shared[i], dout_bt[i]);
        // gradient contribution to weight
        atomicAdd(&dweight_shared[i], norm_bti * dout_bt[i]);
        // gradient contribution to input
        float dval = 0.0f;
        dval += dnorm_i; // term 1
        dval -= dnorm_mean; // term 2
        dval -= norm_bti * dnorm_norm_mean; // term 3
        dval *= rstd_bt; // final scale
        dinp_bt[i] += dval;
    }
    __syncthreads();

    // write to global memory
	for(int i = threadIdx.x; i < C; i+= blockDim.x){
        atomicAdd(&dbias[i], dbias_shared[i]);
        atomicAdd(&dweight[i], dweight_shared[i]);
	}
}

// ----------------------------------------------------------------------------
// kernel launchers

void layernorm_backward1(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C, const int block_size) {
    const int N = B * T;
    const int grid_size = ceil_div(N, block_size);
    layernorm_backward_kernel1<<<grid_size, block_size>>>(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);
}

void layernorm_backward2(float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C, const int block_size) {
    const int N = B * T;
    const int grid_size = ceil_div(32*N, block_size);
    size_t shared_mem_size = 2 * C * sizeof(float);
    layernorm_backward_kernel2<<<grid_size, block_size, shared_mem_size>>>(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);
}

// kernel version dispatch
void layernorm_backward(int kernel_num,
                        float* dinp, float* dweight, float* dbias,
                        const float* dout, const float* inp, const float* weight, const float* mean, const float* rstd,
                        int B, int T, int C,
                        const int block_size) {
    switch (kernel_num) {
        case 1:
            layernorm_backward1(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C, block_size);
            break;
        case 2:
            layernorm_backward2(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C, block_size);
            break;
    default:
            printf("Invalid kernel number\n");
            exit(1);
    }
}

// ----------------------------------------------------------------------------

int main(int argc, char **argv) {
    srand(0);

    int B = 8;
    int T = 1024;
    int C = 768;

    int deviceIdx = 0;
    cudaCheck(hipSetDevice(deviceIdx));

    // first do the forward pass in CPU
    float* out = (float*)malloc(B * T * C * sizeof(float));
    float* mean = (float*)malloc(B * T * sizeof(float));
    float* rstd = (float*)malloc(B * T * sizeof(float));
    float* inp = make_random_float(B * T * C);
    float* weight = make_random_float(C);
    float* bias = make_random_float(C);
    layernorm_forward_cpu(out, mean, rstd, inp, weight, bias, B, T, C);

    // now do the backward pass, again on CPU
    float *dout = make_random_float(B * T * C);
    float *dinp = make_zeros_float(B * T * C);
    float *dweight = make_zeros_float(C);
    float *dbias = make_zeros_float(C);
    layernorm_backward_cpu(dinp, dweight, dbias, dout, inp, weight, mean, rstd, B, T, C);

    // the above calculations act as the reference
    // now let's do the same on the GPU

    // read kernel_num from command line
    int kernel_num = 2;
    if (argc > 1) {
        kernel_num = atoi(argv[1]);
    }
    printf("Using kernel %d\n", kernel_num);

    // move all the variables we need for backward pass onto the GPU
    float* d_dinp;
    float* d_dweight;
    float* d_dbias;
    float* d_dout;
    float* d_inp;
    float* d_weight;
    float* d_mean;
    float* d_rstd;
    cudaCheck(hipMalloc(&d_dinp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dweight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dbias, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_dout, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_inp, B * T * C * sizeof(float)));
    cudaCheck(hipMalloc(&d_weight, C * sizeof(float)));
    cudaCheck(hipMalloc(&d_mean, B * T * sizeof(float)));
    cudaCheck(hipMalloc(&d_rstd, B * T * sizeof(float)));
    // copy over the "inputs" to the backward call
    cudaCheck(hipMemcpy(d_dout, dout, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_inp, inp, B * T * C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_weight, weight, C * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_mean, mean, B * T * sizeof(float), hipMemcpyHostToDevice));
    cudaCheck(hipMemcpy(d_rstd, rstd, B * T * sizeof(float), hipMemcpyHostToDevice));
    // init the "outputs" of the backward call to zeros
    cudaCheck(hipMemset(d_dinp, 0, B * T * C * sizeof(float)));
    cudaCheck(hipMemset(d_dweight, 0, C * sizeof(float)));
    cudaCheck(hipMemset(d_dbias, 0, C * sizeof(float)));

    // launch the kernel
    const int block_size = 256;
    layernorm_backward(kernel_num, d_dinp, d_dweight, d_dbias, d_dout, d_inp, d_weight, d_mean, d_rstd, B, T, C, block_size);

    // check the correctness of the kernel
    printf("Checking correctness...\n");
    printf("dinp:\n");
    validate_result(d_dinp, dinp, "dinp", B * T * C, 1e-3f);
    printf("dweight:\n");
    validate_result(d_dweight, dweight, "dweight", C, 1e-3f);
    printf("dbias:\n");
    validate_result(d_dbias, dbias, "dbias", C, 1e-3f);

    // now time the kernel
    int block_sizes[] = {32, 64, 128, 256, 512, 1024};
    for (int j = 0; j < sizeof(block_sizes) / sizeof(int); j++) {
        int block_size = block_sizes[j];
        int repeat_times = 100;
        float elapsed_time = benchmark_kernel(repeat_times, layernorm_backward, kernel_num,
                                              d_dinp, d_dweight, d_dbias, d_dout, d_inp, d_weight, d_mean, d_rstd,
                                              B, T, C, block_size);
        printf("block_size %4d time %.4f ms\n", block_size, elapsed_time);
    }

    // cleanups
    free(out);
    free(mean);
    free(rstd);
    free(inp);
    free(weight);
    free(bias);
    free(dout);
    free(dinp);
    free(dweight);
    free(dbias);
    cudaCheck(hipFree(d_dinp));
    cudaCheck(hipFree(d_dweight));
    cudaCheck(hipFree(d_dbias));
    cudaCheck(hipFree(d_dout));
    cudaCheck(hipFree(d_inp));
    cudaCheck(hipFree(d_weight));
    cudaCheck(hipFree(d_mean));
    cudaCheck(hipFree(d_rstd));

    return 0;
}
